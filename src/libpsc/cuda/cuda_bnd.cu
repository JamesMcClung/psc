#include "hip/hip_runtime.h"

#include "cuda_iface.h"
#include "cuda_iface_bnd.h"
#include "cuda_bits.h"

#include "psc.h"
#include "psc_fields_cuda.h"
#include "fields.hxx"

#define BLOCKSIZE_X 1
#define BLOCKSIZE_Y 4
#define BLOCKSIZE_Z 4

#define SW (2) // FIXME

template <bool lo, bool hi, typename E>
__global__ static void conducting_wall_H_y(E gt, Int3 ib)
{
  int iz = blockIdx.x * blockDim.x + threadIdx.x - SW;

  auto _d_flds = make_Fields3d<dim_xyz>(gt, ib);

  if (iz >= gt.shape(2) - SW)
    return;

  int my = gt.shape(1) - 2 * SW;

  if (lo) {
    _d_flds(HY, 0, -1, iz) = _d_flds(HY, 0, 1, iz);
    _d_flds(HX, 0, -1, iz) = -_d_flds(HX, 0, 0, iz);
    _d_flds(HZ, 0, -1, iz) = -_d_flds(HZ, 0, 0, iz);
  }

  if (hi) {
    _d_flds(HY, 0, my + 1, iz) = _d_flds(HY, 0, my - 1, iz);
    _d_flds(HX, 0, my, iz) = -_d_flds(HX, 0, my - 1, iz);
    _d_flds(HZ, 0, my, iz) = -_d_flds(HZ, 0, my - 1, iz);
  }
}

template <bool lo, bool hi, typename E>
__global__ static void conducting_wall_E_y(E gt, Int3 ib)
{
  int iz = blockIdx.x * blockDim.x + threadIdx.x - SW;

  auto _d_flds = make_Fields3d<dim_xyz>(gt, ib);

  if (iz >= gt.shape(2) - SW)
    return;

  int my = gt.shape(1) - 2 * SW;

  if (lo) {
    _d_flds(EX, 0, 0, iz) = 0.;
    _d_flds(EX, 0, -1, iz) = _d_flds(EX, 0, 1, iz);
    _d_flds(EY, 0, -1, iz) = -_d_flds(EY, 0, 0, iz);
    _d_flds(EZ, 0, 0, iz) = 0.;
    _d_flds(EZ, 0, -1, iz) = _d_flds(EZ, 0, 1, iz);
  }

  if (hi) {
    _d_flds(EX, 0, my, iz) = 0.;
    _d_flds(EX, 0, my + 1, iz) = _d_flds(EX, 0, my - 1, iz);
    _d_flds(EY, 0, my, iz) = -_d_flds(EY, 0, my - 1, iz);
    _d_flds(EZ, 0, my, iz) = 0.;
    _d_flds(EZ, 0, my + 1, iz) = _d_flds(EZ, 0, my - 1, iz);
  }
}

template <bool lo, bool hi, typename E>
__global__ static void conducting_wall_J_y(E gt, Int3 ib)
{
  int iz = blockIdx.x * blockDim.x + threadIdx.x - SW;

  auto _d_flds = make_Fields3d<dim_xyz>(gt, ib);

  if (iz >= gt.shape(2) - SW)
    return;

  int my = gt.shape(1) - 2 * SW;

  if (lo) {
    _d_flds(JYI, 0, 0, iz) -= _d_flds(JYI, 0, -1, iz);
    _d_flds(JYI, 0, -1, iz) = 0.;
    _d_flds(JXI, 0, 1, iz) += _d_flds(JXI, 0, -1, iz);
    _d_flds(JXI, 0, -1, iz) = 0.;
    _d_flds(JZI, 0, 1, iz) += _d_flds(JZI, 0, -1, iz);
    _d_flds(JZI, 0, -1, iz) = 0.;
  }

  if (hi) {
    _d_flds(JYI, 0, my - 1, iz) -= _d_flds(JYI, 0, my, iz);
    _d_flds(JYI, 0, my, iz) = 0.;
    _d_flds(JXI, 0, my - 1, iz) += _d_flds(JXI, 0, my + 1, iz);
    _d_flds(JXI, 0, my + 1, iz) = 0.;
    _d_flds(JZI, 0, my - 1, iz) += _d_flds(JZI, 0, my + 1, iz);
    _d_flds(JZI, 0, my + 1, iz) = 0.;
  }
}

template <bool lo, bool hi>
static void cuda_conducting_wall_H_y(MfieldsCuda& mflds, int p)
{
  int dimGrid = (mflds.gt().shape(2) + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z;
  conducting_wall_H_y<lo, hi>
    <<<dimGrid, BLOCKSIZE_Z>>>(view_patch(mflds.gt(), p), -mflds.ibn());
  cuda_sync_if_enabled();
}

template <bool lo, bool hi>
static void cuda_conducting_wall_E_y(MfieldsCuda& mflds, int p)
{
  int dimGrid = (mflds.gt().shape(2) + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z;
  conducting_wall_E_y<lo, hi>
    <<<dimGrid, BLOCKSIZE_Z>>>(view_patch(mflds.gt(), p), -mflds.ibn());
  cuda_sync_if_enabled();
}

template <bool lo, bool hi>
static void cuda_conducting_wall_J_y(MfieldsCuda& mflds, int p)
{
  int dimGrid = (mflds.gt().shape(2) + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z;
  conducting_wall_J_y<lo, hi>
    <<<dimGrid, BLOCKSIZE_Z>>>(view_patch(mflds.gt(), p), -mflds.ibn());
  cuda_sync_if_enabled();
}

void cuda_conducting_wall_H_lo_y(MfieldsCuda& mflds, int p)
{
  cuda_conducting_wall_H_y<true, false>(mflds, p);
}

void cuda_conducting_wall_H_hi_y(MfieldsCuda& mflds, int p)
{
  cuda_conducting_wall_H_y<false, true>(mflds, p);
}

void cuda_conducting_wall_H_lo_hi_y(MfieldsCuda& mflds, int p)
{
  cuda_conducting_wall_H_y<true, true>(mflds, p);
}

void cuda_conducting_wall_E_lo_y(MfieldsCuda& mflds, int p)
{
  cuda_conducting_wall_E_y<true, false>(mflds, p);
}

void cuda_conducting_wall_E_hi_y(MfieldsCuda& mflds, int p)
{
  cuda_conducting_wall_E_y<false, true>(mflds, p);
}

void cuda_conducting_wall_E_lo_hi_y(MfieldsCuda& mflds, int p)
{
  cuda_conducting_wall_E_y<true, true>(mflds, p);
}

void cuda_conducting_wall_J_lo_y(MfieldsCuda& mflds, int p)
{
  cuda_conducting_wall_J_y<true, false>(mflds, p);
}

void cuda_conducting_wall_J_hi_y(MfieldsCuda& mflds, int p)
{
  cuda_conducting_wall_J_y<false, true>(mflds, p);
}

void cuda_conducting_wall_J_lo_hi_y(MfieldsCuda& mflds, int p)
{
  cuda_conducting_wall_J_y<true, true>(mflds, p);
}
