#include "hip/hip_runtime.h"

#include "../libpsc/psc_output_fields/fields_item_fields.hxx"
#include "fields_item_dive_cuda.hxx"

#define BND 2
#define BLOCKSIZE_X 1
#define BLOCKSIZE_Y 16
#define BLOCKSIZE_Z 16

// ======================================================================

__global__ static void calc_dive_yz(DFields flds, DFields f, float dy, float dz,
                                    int ldimsy, int ldimsz)
{
  int iy = blockIdx.x * blockDim.x + threadIdx.x;
  int iz = blockIdx.y * blockDim.y + threadIdx.y;

  if (iy >= ldimsy || iz >= ldimsz) {
    return;
  }

  auto _flds = make_Fields3d<dim_xyz>(flds);
  auto _f = make_Fields3d<dim_xyz>(f);
  _f(0, 0, iy, iz) = ((_flds(EY, 0, iy, iz) - _flds(EY, 0, iy - 1, iz)) / dy +
                      (_flds(EZ, 0, iy, iz) - _flds(EZ, 0, iy, iz - 1)) / dz);
}

void cuda_mfields_calc_dive_yz(MfieldsStateCuda& mflds, MfieldsCuda& mf, int p)
{
  auto cmflds = mflds.cmflds();
  auto cmf = mf.cmflds();
  auto dx = mflds.grid().domain.dx;

  int my = mflds.gt().shape(1);
  int mz = mflds.gt().shape(2);

  int grid[2] = {(my + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y,
                 (mz + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z};
  dim3 dimBlock(BLOCKSIZE_Y, BLOCKSIZE_Z);
  dim3 dimGrid(grid[0], grid[1]);

  calc_dive_yz<<<dimGrid, dimBlock>>>((*cmflds)[p], (*cmf)[p], dx[1], dx[2],
                                      cmflds->grid().ldims[1],
                                      cmflds->grid().ldims[2]);
  cuda_sync_if_enabled();
}
